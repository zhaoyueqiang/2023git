#include "hip/hip_runtime.h"
﻿#include<stdio.h>
__global__ void hello_world(void)
{
	printf("GPU: Hello world!\n"); // GPU输出的Hello World!
}
int main(int argc, char** argv)
{
	printf("CPU: Hello world!\n"); // CPU输出的Hello World!
	hello_world << <1, 10 >> > ();
	hipDeviceReset();             // 如果没有这一行就看不到GPU输出的Hello World!
	return 0;
}